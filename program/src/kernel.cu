#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include "kernel.h"

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/* Check for CUDA errors; print and exit if there was a problem.*/
void checkCUDAError(const char *msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

//__global__ void kernCopyPlanetsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
//    int index = threadIdx.x + (blockIdx.x * blockDim.x);
//
//    float c_scale = -1.0f / s_scale;
//
//    if (index < N) {
//        vbo[4 * index + 0] = pos[index].x * c_scale;
//        vbo[4 * index + 1] = pos[index].y * c_scale;
//        vbo[4 * index + 2] = pos[index].z * c_scale;
//        vbo[4 * index + 3] = 1;
//    }
//}

///**
// * Wrapper for call to the kernCopyPlanetsToVBO CUDA kernel.
// */
//void Terrain::copyPlanetsToVBO(float *vbodptr) {
//	dim3 fullBlocksPerGrid((int)ceil(float(numMap) / float(blockSize)));
//
//    kernCopyPlanetsToVBO<<<fullBlocksPerGrid, blockSize>>>(numMap, dev_pos, vbodptr, scene_scale);
//    checkCUDAErrorWithLine("copyPlanetsToVBO failed!");
//
//    hipDeviceSynchronize();
//}

dim3 threadsPerBlock(blockSize);

__global__ void HeightMap(float *height, int pixels)
{
	int octaves_ = octaves, seed_ = seed;
	float amp_ = amplitude, freq_ = frequency;
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < pixels)
		for (int i = 0; i < order; i++){
		octaves_++; amp_ /= 2; freq_ *= 2; seed_++;
		Perlin perl(octaves_, amp_, freq_, seed_);
		height[index] += perl.Get(blockIdx.x, threadIdx.x);
		}
}

void Terrain::MapGen(float* hst_height, unsigned int size, float *time) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	float *dev_height;
	unsigned int numPixels = size;
	dim3 fullBlocksPerGrid((int)ceil(float(numPixels) / blockSize));
		
	hipMalloc((void**)&dev_height, sizeof(hst_height));
	checkCUDAErrorWithLine("hipMalloc dev_height failed!");

	hipMemcpy(hst_height, dev_height, sizeof(hst_height), hipMemcpyHostToDevice);
	checkCUDAErrorWithLine("hipMemcpy hst_height failed!");

	HeightMap << <threadsPerBlock, fullBlocksPerGrid >> >(dev_height, numPixels);

	hipMemcpy(hst_height, dev_height, sizeof(hst_height), hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("hipMemcpy dev_height failed!");

	hipFree(dev_height);
		
	hipEventRecord(stop);	
	hipEventSynchronize(stop);
	hipEventElapsedTime(time, start, stop);
	std::cout << *time << std::endl;
}
