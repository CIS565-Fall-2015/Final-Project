#include "hip/hip_runtime.h"
#include "perlin.h"

#define setup(i,b0,b1,r0,r1)\
	t = vec[i] + N;\
	b0 = ((int)t) & BM;\
	b1 = (b0+1) & BM;\
	r0 = t - (int)t;\
	r1 = r0 - 1.0f;

#define hash(a)\
	a = (a + 0x7ed55d16) + (a << 12);\
	a = (a ^ 0xc761c23c) ^ (a >> 19);\
	a = (a + 0x165667b1) + (a << 5);\
	a = (a + 0xd3a2646c) ^ (a << 9);\
	a = (a + 0xfd7046c5) + (a << 3);\
	a = (a ^ 0xb55a4f09) ^ (a >> 16);

__device__ void Perlin::init_rand(int seed) {
	hash(seed);
	rng = thrust::default_random_engine(float(seed));
	unitDistrib = thrust::uniform_real_distribution<float>(0, 1);
}

__device__ float Perlin::noise1(float arg) {
	int bx0, bx1;
	float rx0, rx1, sx, t, u, v, vec[1];

	vec[0] = arg;

	if (mStart) {
		init_rand(mSeed);
		mStart = false;
		init();
	}

	setup(0, bx0,bx1, rx0,rx1);

	sx = s_curve(rx0);

	u = rx0 * g1[p[bx0]];
	v = rx1 * g1[p[bx1]];

	return lerrp(sx, u, v);
}

__device__ float Perlin::noise2(float vec[2]) {
	int bx0, bx1, by0, by1, b00, b10, b01, b11;
	float rx0, rx1, ry0, ry1, *q, sx, sy, a, b, t, u, v;
	int i, j;

	if (mStart) {
		init_rand(mSeed);
		mStart = false;
		init();
	}

	setup(0,bx0,bx1,rx0,rx1);
	setup(1,by0,by1,ry0,ry1);

	i = p[bx0];
	j = p[bx1];

	b00 = p[i + by0];
	b10 = p[j + by0];
	b01 = p[i + by1];
	b11 = p[j + by1];

	sx = s_curve(rx0);
	sy = s_curve(ry0);

	#define at2(rx,ry) (rx * q[0] + ry * q[1])

	q = g2[b00];
	u = at2(rx0,ry0);
	q = g2[b10];
	v = at2(rx1,ry0);
	a = lerrp(sx, u, v);

	q = g2[b01];
	u = at2(rx0,ry1);
	q = g2[b11];
	v = at2(rx1,ry1);
	b = lerrp(sx, u, v);

	return lerrp(sy, a, b);
}

__device__ float Perlin::noise3(float vec[3]) {
	int bx0, bx1, by0, by1, bz0, bz1, b00, b10, b01, b11;
	float rx0, rx1, ry0, ry1, rz0, rz1, *q, sy, sz, a, b, c, d, t, u, v;
	int i, j;

	if (mStart) {
		init_rand(mSeed);
		mStart = false;
		init();
	}

	setup(0, bx0,bx1, rx0,rx1);
	setup(1, by0,by1, ry0,ry1);
	setup(2, bz0,bz1, rz0,rz1);

	i = p[bx0];
	j = p[bx1];

	b00 = p[i + by0];
	b10 = p[j + by0];
	b01 = p[i + by1];
	b11 = p[j + by1];

	t  = s_curve(rx0);
	sy = s_curve(ry0);
	sz = s_curve(rz0);

    #define at3(rx,ry,rz) ( rx * q[0] + ry * q[1] + rz * q[2] )

	q = g3[b00 + bz0] ; u = at3(rx0,ry0,rz0);
	q = g3[b10 + bz0] ; v = at3(rx1,ry0,rz0);
	a = lerrp(t, u, v);

	q = g3[b01 + bz0] ; u = at3(rx0,ry1,rz0);
	q = g3[b11 + bz0] ; v = at3(rx1,ry1,rz0);
	b = lerrp(t, u, v);

	c = lerrp(sy, a, b);

	q = g3[b00 + bz1] ; u = at3(rx0,ry0,rz1);
	q = g3[b10 + bz1] ; v = at3(rx1,ry0,rz1);
	a = lerrp(t, u, v);

	q = g3[b01 + bz1] ; u = at3(rx0,ry1,rz1);
	q = g3[b11 + bz1] ; v = at3(rx1,ry1,rz1);
	b = lerrp(t, u, v);

	d = lerrp(sy, a, b);

	return lerrp(sz, c, d);
}

__device__ void Perlin::normalize2(float v[2]) {
	float s;

	s = (float)sqrt(v[0] * v[0] + v[1] * v[1]);
	s = 1.0f/s;
	v[0] = v[0] * s;
	v[1] = v[1] * s;
}

__device__ void Perlin::normalize3(float v[3]) {
	float s;

	s = (float)sqrt(v[0] * v[0] + v[1] * v[1] + v[2] * v[2]);
	s = 1.0f/s;

	v[0] = v[0] * s;
	v[1] = v[1] * s;
	v[2] = v[2] * s;
}

__device__ void Perlin::init(void) {
	int i, j, k;

	for (i = 0 ; i < B ; i++) {
		p[i] = i;
		g1[i] = (float)((int(unitDistrib(rng)*(float)RAND_MAX) % (B + B)) - B) / B;
		for (j = 0 ; j < 2 ; j++) {
			g2[i][j] = (float)((int(unitDistrib(rng)*(float)RAND_MAX) % (B + B)) - B) / B;
		}
		normalize2(g2[i]);
		for (j = 0 ; j < 3 ; j++) {
			g3[i][j] = (float)((int(unitDistrib(rng)*(float)RAND_MAX) % (B + B)) - B) / B;
		}
		normalize3(g3[i]);
	}

	while (--i) {
		k = p[i];
		p[i] = p[j = int(unitDistrib(rng)*(float)RAND_MAX) % B];
		p[j] = k;
	}

	for (i = 0 ; i < B + 2 ; i++) {
		p[B + i] = p[i];
		g1[B + i] = g1[i];
		for (j = 0 ; j < 2 ; j++) {
			g2[B + i][j] = g2[i][j];
		}
		for (j = 0 ; j < 3 ; j++) {
			g3[B + i][j] = g3[i][j];
		}
	}
}


__device__ float Perlin::perlin_noise_2D(float vec[2]) {
	int terms = mOctaves;
	float freq = mFrequency;
	float result = 0.0f;
	float amp = mAmplitude;
	
	vec[0] *= freq;
	vec[1] *= freq;
	
	for(int i=0; i < terms; i++) {
		result += noise2(vec)*amp;
		vec[0] *= 2.0f;
		vec[1] *= 2.0f;
		amp*=0.5f;
	}
	return result;
}

__device__ float Perlin::perlin_noise_3D(float vec[3]) {
	int terms = mOctaves;
	float freq = mFrequency;
	float result = 0.0f;
	float amp = mAmplitude;

	vec[0] *= freq;
	vec[1] *= freq;
	vec[2] *= freq;

	for(int i=0; i<terms; i++ ) {
		result += noise3(vec)*amp;
		vec[0] *= 2.0f;
		vec[1] *= 2.0f;
		vec[2] *= 2.0f;
		amp*=0.5f;
	}
	return result;
}

__device__ Perlin::Perlin(int octaves,float freq,float amp,int seed) {
	mOctaves = octaves;
	mFrequency = freq;
	mAmplitude = amp;
	mSeed = seed;
	mStart = true;
}